#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <iomanip>
//CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
using namespace std;

//calculate offset from row,col and ld in row-major matrix,ld is the width of the matrix
__device__ __forceinline__ int OFFSET(int row,int col,int ld)
{
    return row*ld+col;
}

//transfer float4
__device__ __forceinline__  float4& FETCH_FLOAT4(float &pointer)
{
    return reinterpret_cast<float4*>(&pointer)[0];
}

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}
//gemm_v1
__global__ void gemm_v1( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    //matrix C's index
    int row=blockIdx.y*blockDim.y+threadIdx.y;
    int col=blockIdx.x*blockDim.x+threadIdx.x;
    if(row<M&&col<N)
    {
        float sum=0.0f;
        for(int k=0;k<K;++k)
        {
            sum+=A[row*K+k]*B[k*N+col];
        }
        C[row*N+col]=sum;
    }
}

//gemm_v1_alter
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X  // width of block of C that each thread calculate
    > 
__global__ void gemm_v1_alter( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    

    //register for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X]={0};



    for(int tile_idx=0;tile_idx<K;tile_idx+=BLOCK_SIZE_K)
    {
        #pragma unroll
        for(int j=0;j<BLOCK_SIZE_K;++j)
        {
            #pragma unroll
            for(int thread_y=0;thread_y<THREAD_SIZE_Y;++thread_y)
            {
                #pragma unroll
                for(int thread_x=0;thread_x<THREAD_SIZE_X;++thread_x)
                {                 
                    accum[thread_y][thread_x]+=A[OFFSET(
                                            BLOCK_SIZE_M*by+THREAD_SIZE_Y*ty+thread_y, // row
                                            tile_idx+j, // col
                                            K )]*
                                            B[OFFSET(
                                            tile_idx+j, // row
                                            BLOCK_SIZE_N*bx+THREAD_SIZE_X*tx+thread_x, // col
                                            N )];

                }
            }
        }
    }

    // // store back to C
    // #pragma unroll
    // for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) 
    // {
    //     #pragma unroll
    //     for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x+=4) 
    //     {
    //         FETCH_FLOAT4(C[OFFSET(
    //             BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
    //             BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
    //             N)]) = FETCH_FLOAT4(accum[thread_y][thread_x]);
    //     }
    // }

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) 
    {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) 
        {
            C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)]= accum[thread_y][thread_x];
        }
    }
}

//gemm_v2
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N  // width of block of C that each thread block calculate
    > 
__global__ void gemm_v2( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_NUM_PER_BLOCK = BLOCK_SIZE_M * BLOCK_SIZE_N;

    // thread id in cur Block
    const int tid = ty * BLOCK_SIZE_M + tx;

    //register for C
    float accum=0;

    // shared memory
    __shared__ float As[BLOCK_SIZE_M][BLOCK_SIZE_K];
    __shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K ;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N ;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW ; 
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW ;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    for(int tile_idx=0;tile_idx<K;tile_idx+=BLOCK_SIZE_K)
    {
        
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
            // printf("%d %d\n",A_TILE_ROW_START,A_TILE_COL);
            As[A_TILE_ROW_START + i][A_TILE_COL]= A[OFFSET(
                A_TILE_ROW_START + i, // row
                A_TILE_COL + tile_idx, // col
                M )];  
        }
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
            Bs[B_TILE_ROW_START + i][B_TILE_COL] = B[OFFSET(
                tile_idx + B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )];
        }
         
        __syncthreads();
        #pragma unroll
        for(int j=0;j<BLOCK_SIZE_K;++j)
        {           
            accum+=As[ty][j]*Bs[j][tx];   
            // printf("%f %f %f\n",accum,As[tx][j],Bs[j][ty]);       
        }
    }

C[OFFSET(
    BLOCK_SIZE_M * by + ty,
    BLOCK_SIZE_N * bx + tx,
    N)] +=accum; 
}

//gemm_v3
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X  // width of block of C that each thread calculate
    > 
__global__ void gemm_v3( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;


    //register for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X]={0};

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[BLOCK_SIZE_M][BLOCK_SIZE_K];
    __shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; 
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    for(int tile_idx=0;tile_idx<K;tile_idx+=BLOCK_SIZE_K)
    {
        //load current tile from global mem
        // printf("%d %d %d\n",A_TILE_ROW_START,A_TILE_COL,OFFSET(
        //     A_TILE_ROW_START, 
        //     A_TILE_COL + tile_idx, 
        //     K ));
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
            FETCH_FLOAT4(As[A_TILE_ROW_START + i][A_TILE_COL]) = FETCH_FLOAT4(A[OFFSET(
                A_TILE_ROW_START + i, // row
                A_TILE_COL + tile_idx, // col
                K )]);
                
        }
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
            FETCH_FLOAT4(Bs[B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                tile_idx + B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
        }
        __syncthreads();
        #pragma unroll
        for(int j=0;j<BLOCK_SIZE_K;++j)
        {
            #pragma unroll
            for(int thread_y=0;thread_y<THREAD_SIZE_Y;++thread_y)
            {
                #pragma unroll
                for(int thread_x=0;thread_x<THREAD_SIZE_X;++thread_x)
                {
                    accum[thread_y][thread_x]+=As[THREAD_SIZE_Y*ty+thread_y][j]*Bs[j][THREAD_SIZE_X*tx+thread_x];
                }
            }
        }
    }

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) 
    {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x+=4) 
        {
            FETCH_FLOAT4(C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)]) = FETCH_FLOAT4(accum[thread_y][thread_x]);
        }
    }
}

//gemm_v4
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X  // width of block of C that each thread calculate
    > 
__global__ void gemm_v4( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;


    //register for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X]={0};

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[BLOCK_SIZE_K][BLOCK_SIZE_M];
    __shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];


    // registers load global memory
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4*ldg_num_a];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; 
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    for(int tile_idx=0;tile_idx<K;tile_idx+=BLOCK_SIZE_K)
    {
        //load current tile from global mem
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
            int ldg_index = i / A_TILE_ROW_STRIDE * 4;
            FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                A_TILE_ROW_START + i, // row
                A_TILE_COL + tile_idx, // col
                K )]);
            As[A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
            As[A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
            As[A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
            As[A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
                
        }
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
            FETCH_FLOAT4(Bs[B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                tile_idx + B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
        }
        __syncthreads();
        #pragma unroll
        for(int j=0;j<BLOCK_SIZE_K;++j)
        {
            #pragma unroll
            for(int thread_y=0;thread_y<THREAD_SIZE_Y;++thread_y)
            {
                #pragma unroll
                for(int thread_x=0;thread_x<THREAD_SIZE_X;++thread_x)
                {
                    accum[thread_y][thread_x]+=As[j][THREAD_SIZE_Y*ty+thread_y]*Bs[j][THREAD_SIZE_X*tx+thread_x];
                }
            }
        }
    }

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) 
    {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x+=4) 
        {
            FETCH_FLOAT4(C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)]) = FETCH_FLOAT4(accum[thread_y][thread_x]);
        }
    }
}

//gemm_v5
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X  // width of block of C that each thread calculate
    > 
__global__ void gemm_v5( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;


    //register for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X]={0};

    // registers for A and B
    float frag_a[THREAD_SIZE_Y];
    float frag_b[THREAD_SIZE_X];

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[BLOCK_SIZE_M][BLOCK_SIZE_K];
    __shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; 
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    for(int tile_idx=0;tile_idx<K;tile_idx+=BLOCK_SIZE_K)
    {
        //load current tile from global mem
        // printf("%d %d %d\n",A_TILE_ROW_START,A_TILE_COL,OFFSET(
        //     A_TILE_ROW_START, 
        //     A_TILE_COL + tile_idx, 
        //     K ));
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
            FETCH_FLOAT4(As[A_TILE_ROW_START + i][A_TILE_COL]) = FETCH_FLOAT4(A[OFFSET(
                A_TILE_ROW_START + i, // row
                A_TILE_COL + tile_idx, // col
                K )]);
                
        }
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
            FETCH_FLOAT4(Bs[B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                tile_idx + B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
        }
        __syncthreads();


        #pragma unroll
        for(int j=0;j<BLOCK_SIZE_K;++j)
        {
            // load A from shared memory to register
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y++) {
                frag_a[thread_y] = As[THREAD_SIZE_Y * ty + thread_y][j];
            }
            // load B from shared memory to register
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
                FETCH_FLOAT4(frag_b[thread_x]) = FETCH_FLOAT4(Bs[j][THREAD_SIZE_X * tx + thread_x]);
            }

            #pragma unroll
            for(int thread_y=0;thread_y<THREAD_SIZE_Y;++thread_y)
            {
                #pragma unroll
                for(int thread_x=0;thread_x<THREAD_SIZE_X;++thread_x)
                {    
                    accum[thread_y][thread_x]+=frag_a[thread_y]* frag_b[thread_x] ;
                }
            }
        }
    }

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) 
    {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x+=4) 
        {
            FETCH_FLOAT4(C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)]) = FETCH_FLOAT4(accum[thread_y][thread_x]);
        }
    }
}

//gemm_v6
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X  // width of block of C that each thread calculate
    > 
__global__ void gemm_v6( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;


    //register for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X]={0};

    // registers for A and B
    float frag_a[THREAD_SIZE_Y];
    float frag_b[THREAD_SIZE_X];

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[BLOCK_SIZE_K][BLOCK_SIZE_M];
    __shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];


    // registers load global memory
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4*ldg_num_a];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; 
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    for(int tile_idx=0;tile_idx<K;tile_idx+=BLOCK_SIZE_K)
    {
        //load current tile from global mem
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
            int ldg_index = i / A_TILE_ROW_STRIDE * 4;
            FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                A_TILE_ROW_START + i, // row
                A_TILE_COL + tile_idx, // col
                K )]);
            As[A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
            As[A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
            As[A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
            As[A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
                
        }
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
            FETCH_FLOAT4(Bs[B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                tile_idx + B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
        }
        __syncthreads();


        #pragma unroll
        for(int j=0;j<BLOCK_SIZE_K;++j)
        {
            // load A from shared memory to register
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y+=4) {
                FETCH_FLOAT4(frag_a[thread_y]) = FETCH_FLOAT4(As[j][THREAD_SIZE_Y * ty + thread_y]);
            }
            // load B from shared memory to register
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
                FETCH_FLOAT4(frag_b[thread_x]) = FETCH_FLOAT4(Bs[j][THREAD_SIZE_X * tx + thread_x]);
            }

            #pragma unroll
            for(int thread_y=0;thread_y<THREAD_SIZE_Y;++thread_y)
            {
                #pragma unroll
                for(int thread_x=0;thread_x<THREAD_SIZE_X;++thread_x)
                {    
                    accum[thread_y][thread_x]+=frag_a[thread_y]* frag_b[thread_x] ;
                }
            }
        }
    }

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) 
    {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x+=4) 
        {
            FETCH_FLOAT4(C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)]) = FETCH_FLOAT4(accum[thread_y][thread_x]);
        }
    }
}

//gemm_v7
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X  // width of block of C that each thread calculate
    > 
__global__ void gemm_v7( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[2][BLOCK_SIZE_M][BLOCK_SIZE_K];
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};
    // registers for A and B
    float frag_a[2][THREAD_SIZE_Y];
    float frag_b[2][THREAD_SIZE_X];
    // registers load global memory
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4*ldg_num_a];
    float ldg_b_reg[4*ldg_num_b];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; 
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    //transfer first tile from global mem to shared mem
    // load A from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(As[0][A_TILE_ROW_START + i][A_TILE_COL]) = FETCH_FLOAT4(A[OFFSET(
            A_TILE_ROW_START + i, // row
            A_TILE_COL, // col
            K )]);
    }
    // load B from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(Bs[0][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
    }
    __syncthreads();
    // load A from shared memory to register
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y ++) {
        frag_a[0][thread_y] = As[0][THREAD_SIZE_Y * ty + thread_y][0];
    }
    // load B from shared memory to register
    #pragma unroll
    for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
        FETCH_FLOAT4(frag_b[0][thread_x]) = FETCH_FLOAT4(Bs[0][0][THREAD_SIZE_X * tx + thread_x]);
    }

    int write_stage_idx = 1;
    int tile_idx = 0;
    do{
        tile_idx += BLOCK_SIZE_K;
        // load next tile from global mem
        if(tile_idx< K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL + tile_idx, // col
                    K )]);
            }
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_b_reg[ldg_index]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL, // col
                    N )]);
            }
        }

        int load_stage_idx = write_stage_idx ^ 1;

        #pragma unroll
        for(int j=0; j<BLOCK_SIZE_K-1; ++j){
            // load next tile from shared mem to register 
            // load A from shared memory to register
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y ++) {
                frag_a[(j+1)%2][thread_y] = As[load_stage_idx][THREAD_SIZE_Y * ty + thread_y][j+1];
            }
            // load B from shared memory to register
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
                FETCH_FLOAT4(frag_b[(j+1)%2][thread_x]) = FETCH_FLOAT4(Bs[load_stage_idx][j+1][THREAD_SIZE_X * tx + thread_x]);
            }
            // compute C THREAD_SIZE_X x THREAD_SIZE_Y
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[j%2][thread_y] * frag_b[j%2][thread_x];
                }
            }
        }

        if(tile_idx < K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(As[write_stage_idx][A_TILE_ROW_START + i][A_TILE_COL]) = FETCH_FLOAT4(ldg_a_reg[ldg_index]);
            }
            // load B from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(ldg_b_reg[ldg_index]);
            }
            // use double buffer, only need one sync
            __syncthreads();
            // switch
            write_stage_idx ^= 1;
        }

        // load first tile from shared mem to register of next iter
        // load A from shared memory to register
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y ++) {
            frag_a[0][thread_y] =As[load_stage_idx^1][THREAD_SIZE_Y * ty + thread_y][0];
        }
        // load B from shared memory to register
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
            FETCH_FLOAT4(frag_b[0][thread_x]) = FETCH_FLOAT4(Bs[load_stage_idx^1][0][THREAD_SIZE_X * tx + thread_x]);
        }
        //compute last tile mma THREAD_SIZE_X x THREAD_SIZE_Y
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                accum[thread_y][thread_x] += frag_a[1][thread_y] * frag_b[1][thread_x];
            }
        }
    }while(tile_idx< K);

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x+=4) {
            FETCH_FLOAT4(C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)]) = FETCH_FLOAT4(accum[thread_y][thread_x]);
        }
    }
}

//gemm_v8
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X // width of block of C that each thread calculate
    > 
__global__ void gemm_v8( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};
    // registers for A and B
    float frag_a[2][THREAD_SIZE_Y];
    float frag_b[2][THREAD_SIZE_X];
    // registers load global memory
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4*ldg_num_a];
    float ldg_b_reg[4*ldg_num_b];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; 
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    //transfer first tile from global mem to shared mem
    // load A from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
        int ldg_index = i / A_TILE_ROW_STRIDE * 4;
        FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
            A_TILE_ROW_START + i, // row
            A_TILE_COL, // col
            K )]);
        As[0][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
        As[0][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
        As[0][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
        As[0][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
    }
    // load B from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(Bs[0][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
    }
    __syncthreads();
    // load A from shared memory to register
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
        FETCH_FLOAT4(frag_a[0][thread_y]) = FETCH_FLOAT4(As[0][0][THREAD_SIZE_Y * ty + thread_y]);
    }
    // load B from shared memory to register
    #pragma unroll
    for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
        FETCH_FLOAT4(frag_b[0][thread_x]) = FETCH_FLOAT4(Bs[0][0][THREAD_SIZE_X * tx + thread_x]);
    }

    int write_stage_idx = 1;
    int tile_idx = 0;
    do{
        tile_idx += BLOCK_SIZE_K;
        // load next tile from global mem
        if(tile_idx< K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL + tile_idx, // col
                    K )]);
            }
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_b_reg[ldg_index]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL, // col
                    N )]);
            }
        }

        int load_stage_idx = write_stage_idx ^ 1;

        #pragma unroll
        for(int j=0; j<BLOCK_SIZE_K-1; ++j){
            // load next tile from shared mem to register 
            // load A from shared memory to register
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
                FETCH_FLOAT4(frag_a[(j+1)%2][thread_y]) = FETCH_FLOAT4(As[load_stage_idx][j+1][THREAD_SIZE_Y * ty + thread_y]);
            }
            // load B from shared memory to register
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
                FETCH_FLOAT4(frag_b[(j+1)%2][thread_x]) = FETCH_FLOAT4(Bs[load_stage_idx][j+1][THREAD_SIZE_X * tx + thread_x]);
            }
            // compute C THREAD_SIZE_X x THREAD_SIZE_Y
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[j%2][thread_y] * frag_b[j%2][thread_x];
                }
            }
        }

        if(tile_idx < K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                As[write_stage_idx][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
                As[write_stage_idx][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
                As[write_stage_idx][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
                As[write_stage_idx][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
            }
            // load B from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(ldg_b_reg[ldg_index]);
            }
            // use double buffer, only need one sync
            __syncthreads();
            // switch
            write_stage_idx ^= 1;
        }

        // load first tile from shared mem to register of next iter
        // load A from shared memory to register
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
            FETCH_FLOAT4(frag_a[0][thread_y]) = FETCH_FLOAT4(As[load_stage_idx^1][0][THREAD_SIZE_Y * ty + thread_y]);
        }
        // load B from shared memory to register
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
            FETCH_FLOAT4(frag_b[0][thread_x]) = FETCH_FLOAT4(Bs[load_stage_idx^1][0][THREAD_SIZE_X * tx + thread_x]);
        }
        //compute last tile mma THREAD_SIZE_X x THREAD_SIZE_Y
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                accum[thread_y][thread_x] += frag_a[1][thread_y] * frag_b[1][thread_x];
            }
        }
    }while(tile_idx< K);

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x+=4) {
            FETCH_FLOAT4(C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)]) = FETCH_FLOAT4(accum[thread_y][thread_x]);
        }
    }
}

int main(int argc, char** argv) {
    if (argc != 4) {
        printf("usage: ./main [M] [K] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t K = atoi(argv[2]);
    size_t N = atoi(argv[3]);

    assert( M%8 == 0); 
    assert( N%8 == 0); 
    assert( K%8 == 0); 

    size_t bytes_A = sizeof(float) * M * K;
    size_t bytes_B = sizeof(float) * K * N;
    size_t bytes_C = sizeof(float) * M * N;
    float* h_A = (float*)malloc(bytes_A);
    float* h_B = (float*)malloc(bytes_B);
    float* h_C = (float*)malloc(bytes_C);
    float* h_C1 = (float*)malloc(bytes_C);

    float* d_A;
    float* d_B;
    float* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_B, bytes_B));
    checkCudaErrors(hipMalloc(&d_C, bytes_C));

    const int BLOCK_SIZE_M = 128;
    const int BLOCK_SIZE_K = 8;
    const int BLOCK_SIZE_N = 128;
    const int THREAD_SIZE_X = 8;
    const int THREAD_SIZE_Y = 8;

    // generate A
    for( int i = 0; i < M * K; i++ ){
        h_A[i] = i / 13;
    }

    // generate B
    for( int i = 0; i < K * N; i++ ) {
        h_B[i] = i % 13;
    }

    checkCudaErrors(hipMemcpy( d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_B, h_B, bytes_B, hipMemcpyHostToDevice));
    
    int nIter = 1;
    checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        gemm_v1<<< dim3(N/16,M/16), dim3(16,16) >>>(d_A, d_B, d_C, M, N, K);
        gemm_v1_alter<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
        gemm_v2<16,16,16> 
        <<< dim3(N/16,M/16), dim3(16,16) >>>(d_A, d_B, d_C, M, N, K);
        gemm_v3<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
        gemm_v4<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
        gemm_v5<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
        gemm_v6<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
        gemm_v7<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
        gemm_v8<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
    }


    checkCudaErrors(hipMemcpy( h_C, d_C, bytes_C, hipMemcpyDeviceToHost));

    // cublas
    hipblasHandle_t blas_handle;  
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    for (int run = 0 ; run < nIter; run ++ ) {
        hipblasSgemm (blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
            M, N, K, &alpha, 
            d_A, K, d_B, N, &beta, d_C, N
        );
    }


    checkCudaErrors(hipMemcpy( h_C1, d_C, bytes_C, hipMemcpyDeviceToHost))
    hipblasDestroy(blas_handle); 
    
    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
}
